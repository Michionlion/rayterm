#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "payload.h"
#include "random.h"

// attribute program
// very slightly inspired from OptiX SDK example 'optixGeometryTriangles'
// (there's literally no documentation on GeometryTriangles)

rtDeclareVariable(rtObject, sysRootObject, , );
rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(float, distance, rtIntersectionDistance, );

rtBuffer<float3> vbuf;
rtBuffer<float3> nbuf;
rtBuffer<int3> vidx;
rtBuffer<int3> nidx;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayPayload, thePayload, rtPayload, );

rtDeclareVariable(optix::float3, varAttenuation, , );
rtDeclareVariable(float, varRoughness, , );

RT_PROGRAM void closest_hit() {
    thePayload.depth -= 1;
    if (thePayload.depth > 0) {
        // calculate interpolated vertex (shading) normal
        const optix::int3 n_idx = nidx[rtGetPrimitiveIndex()];
        const optix::float3 normal =
            optix::normalize(nbuf[n_idx.y] * barycentrics.x + nbuf[n_idx.z] * barycentrics.y +
                             nbuf[n_idx.x] * (1.0f - barycentrics.x - barycentrics.y));

        optix::float3 reflect_dir = optix::reflect(theRay.direction, normal);
        optix::float3 random_dir = optix::normalize(random_in_uhemisphere(thePayload.rand, normal));
        optix::float3 ref_dir    = optix::lerp(reflect_dir, random_dir, varRoughness);
        // should test to see if ref_dir is below geometry with geometric normal -- probably doesn't
        // really matter
        optix::float3 hit_pos = theRay.origin + distance * theRay.direction;
        optix::Ray bounce     = optix::make_Ray(hit_pos, ref_dir, 0, 0.00001f, RT_DEFAULT_MAX);

        rtTrace(sysRootObject, bounce, thePayload);
        thePayload.radiance = varAttenuation * thePayload.radiance;
    } else {
        thePayload.radiance = optix::make_float3(0);
    }
}

RT_PROGRAM void any_hit() {}
