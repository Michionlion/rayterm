#include "hip/hip_runtime.h"
#include "config.h"

#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "payload.h"
#include "random.h"

// attribute program
// very slightly inspired from OptiX SDK example 'optixGeometryTriangles'
// (there's literally no documentation on GeometryTriangles)

rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );

rtDeclareVariable(float, distance, rtIntersectionDistance, );

rtBuffer<float3> vbuf;
rtBuffer<float3> nbuf;
rtBuffer<int3> vidx;
rtBuffer<int3> nidx;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayPayload, thePayload, rtPayload, );

rtDeclareVariable(optix::float3, varColor, varColor, );

RT_PROGRAM void closest_hit() {
    const optix::int3 v_idx = vidx[rtGetPrimitiveIndex()];
    const optix::float3 v0  = vbuf[v_idx.x];
    const optix::float3 v1  = vbuf[v_idx.y];
    const optix::float3 v2  = vbuf[v_idx.z];
    // const optix::float3 Ng        = optix::cross(v1 - v0, v2 - v0);
    // const float3 geometric_normal = optix::normalize(Ng);

    const optix::int3 n_idx = nidx[rtGetPrimitiveIndex()];
    const optix::float3 shading_normal =
        optix::normalize(nbuf[n_idx.y] * barycentrics.x + nbuf[n_idx.z] * barycentrics.y +
                         nbuf[n_idx.x] * (1.0f - barycentrics.x - barycentrics.y));

    // optix::float3 dir = random_in_uhemisphere(thePayload.rand, geometric_normal);
    // thePayload.rand   = dir.z;

    // thePayload.radiance =
    //     make_float3(barycentrics.x, barycentrics.y, 1.0f - barycentrics.x - barycentrics.y);
    thePayload.radiance = shading_normal * 0.5f + 0.5f;
}

RT_PROGRAM void any_hit() {}
