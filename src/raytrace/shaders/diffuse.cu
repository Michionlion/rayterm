#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "payload.h"
#include "random.h"

// attribute program
// very slightly inspired from OptiX SDK example 'optixGeometryTriangles'
// (there's literally no documentation on GeometryTriangles)

rtDeclareVariable(rtObject, sysRootObject, , );
rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(float, distance, rtIntersectionDistance, );

rtBuffer<float3> vbuf;
rtBuffer<float3> nbuf;
rtBuffer<int3> vidx;
rtBuffer<int3> nidx;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayPayload, thePayload, rtPayload, );

rtDeclareVariable(optix::float3, varAttenuation, , );

RT_PROGRAM void closest_hit() {
    thePayload.depth -= 1;
    if (thePayload.depth > 0) {
        const optix::int3 v_idx = vidx[rtGetPrimitiveIndex()];
        const optix::float3 v0  = vbuf[v_idx.x];
        const optix::float3 v1  = vbuf[v_idx.y];
        const optix::float3 v2  = vbuf[v_idx.z];

        // calculate geometric (face) normal
        // const optix::float3 Ng        = optix::cross(v1 - v0, v2 - v0);
        // const float3 normal = optix::normalize(Ng);

        // calculate interpolated vertex (shading) normal
        const optix::int3 n_idx = nidx[rtGetPrimitiveIndex()];
        const optix::float3 normal =
            optix::normalize(nbuf[n_idx.y] * barycentrics.x + nbuf[n_idx.z] * barycentrics.y +
                             nbuf[n_idx.x] * (1.0f - barycentrics.x - barycentrics.y));

        optix::float3 ref_dir = optix::normalize(random_in_uhemisphere(thePayload.rand, normal));
        optix::float3 hit_pos = theRay.origin + distance * theRay.direction;
        optix::Ray bounce     = optix::make_Ray(hit_pos, ref_dir, 0, 0.001f, RT_DEFAULT_MAX);
        rtTrace(sysRootObject, bounce, thePayload);
        thePayload.radiance = varAttenuation * thePayload.radiance;
    } else {
        thePayload.radiance = optix::make_float3(0);
    }
}

RT_PROGRAM void any_hit() {}
