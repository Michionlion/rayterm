#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "payload.h"
#include "random.h"

// attribute program
// very slightly inspired from OptiX SDK example 'optixGeometryTriangles'
// (there's literally no documentation on GeometryTriangles)

rtDeclareVariable(rtObject, sysRootObject, , );
rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(float, distance, rtIntersectionDistance, );

rtBuffer<float3> vbuf;
rtBuffer<float3> nbuf;
rtBuffer<int3> vidx;
rtBuffer<int3> nidx;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayPayload, thePayload, rtPayload, );

rtDeclareVariable(optix::float3, varAttenuation, , );
rtDeclareVariable(float, varIndexOfRefraction, , );

RT_PROGRAM void closest_hit() {
    thePayload.depth -= 1;
    if (thePayload.depth > 0) {
        // calculate interpolated vertex (shading) normal
        const optix::int3 n_idx = nidx[rtGetPrimitiveIndex()];
        const optix::float3 normal =
            optix::normalize(nbuf[n_idx.y] * barycentrics.x + nbuf[n_idx.z] * barycentrics.y +
                             nbuf[n_idx.x] * (1.0f - barycentrics.x - barycentrics.y));

        optix::Ray bounce;
        optix::float3 outward_normal;
        optix::float3 reflected = optix::reflect(theRay.direction, normal);
        float ni_over_nt;
        optix::float3 refracted;
        float reflect_prob;
        float cosine;

        if (optix::dot(theRay.direction, normal) > 0.0f) {
            outward_normal = -normal;
            ni_over_nt     = varIndexOfRefraction;
            cosine         = optix::dot(theRay.direction, normal) / optix::length(theRay.direction);
            cosine         = sqrtf(
                1.0f - varIndexOfRefraction * varIndexOfRefraction * (1.0f - cosine * cosine));
        } else {
            outward_normal = normal;
            ni_over_nt     = 1.0f / varIndexOfRefraction;
            cosine = -optix::dot(theRay.direction, normal) / optix::length(theRay.direction);
        }

        // do refraction (optix does something weird)
        optix::float3 incoming = optix::normalize(theRay.direction);
        float dt               = optix::dot(incoming, outward_normal);
        float discriminant     = 1.0f - ni_over_nt * ni_over_nt * (1 - dt * dt);
        if (discriminant > 0.0f) {
            refracted = ni_over_nt * (incoming - outward_normal * dt) -
                        outward_normal * sqrtf(discriminant);

            // calculate schlick approximation
            float r0     = (1.0f - varIndexOfRefraction) / (1.0f + varIndexOfRefraction);
            r0           = r0 * r0;
            reflect_prob = r0 + (1.0f - r0) * pow((1.0f - cosine), 5.0f);

        } else {
            reflect_prob = 1.0f;
        }

        optix::float3 hit_pos = theRay.origin + distance * theRay.direction;
        if (thePayload.rand.get() < reflect_prob) {
            bounce = optix::make_Ray(hit_pos, reflected, 0, 0.00001f, RT_DEFAULT_MAX);
        } else {
            bounce = optix::make_Ray(hit_pos, refracted, 0, 0.00001f, RT_DEFAULT_MAX);
        }

        rtTrace(sysRootObject, bounce, thePayload);
        thePayload.radiance = varAttenuation * thePayload.radiance;
    } else {
        thePayload.radiance = optix::make_float3(0);
    }
}

RT_PROGRAM void any_hit() {}
