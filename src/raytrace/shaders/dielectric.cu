#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "payload.h"
#include "random.h"

// attribute program
// very slightly inspired from OptiX SDK example 'optixGeometryTriangles'
// (there's literally no documentation on GeometryTriangles)

rtDeclareVariable(rtObject, sysRootObject, , );
rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(float, distance, rtIntersectionDistance, );

rtBuffer<float3> vbuf;
rtBuffer<float3> nbuf;
rtBuffer<int3> vidx;
rtBuffer<int3> nidx;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayPayload, thePayload, rtPayload, );

rtDeclareVariable(optix::float3, varAttenuation, , );
rtDeclareVariable(float, varIndexOfRefraction, , );

RT_PROGRAM void closest_hit() {
    thePayload.depth -= 1;
    if (thePayload.depth > 0) {
        // calculate interpolated vertex (shading) normal
        const optix::int3 n_idx = nidx[rtGetPrimitiveIndex()];
        const optix::float3 normal =
            optix::normalize(nbuf[n_idx.y] * barycentrics.x + nbuf[n_idx.z] * barycentrics.y +
                             nbuf[n_idx.x] * (1.0f - barycentrics.x - barycentrics.y));
        optix::Ray bounce;

        optix::float3 outward_normal;
        optix::float3 reflected = optix::reflect(r_in.direction, normal);
        float ni_over_nt;
        optix::float3 refracted;
        float reflect_prob;
        float cosine;

        if (optix::dot(theRay.direction, normal) > 0.f) {
            outward_normal = -normal;
            ni_over_nt     = ref_idx;
            cosine         = dot(r_in.direction, hit_rec_normal) / vec3f(r_in.direction).length();
            cosine         = sqrtf(1.f - ref_idx * ref_idx * (1.f - cosine * cosine));
        } else {
            outward_normal = hit_rec_normal;
            ni_over_nt     = 1.0 / ref_idx;
            cosine         = -dot(r_in.direction, hit_rec_normal) / vec3f(r_in.direction).length();
        }
        if (refract(r_in.direction, outward_normal, ni_over_nt, refracted))
            reflect_prob = schlick(cosine, ref_idx);
        else
            reflect_prob = 1.f;
        optix::float3 hit_pos = theRay.origin + distance * theRay.direction;
        if (rnd() < reflect_prob)
            bounce = optix::make_Ray(hit_pos, reflected, 0, 0.00001f, RT_DEFAULT_MAX);
        else
            bounce = optix::make_Ray(hit_pos, refracted, 0, 0.00001f, RT_DEFAULT_MAX);

        rtTrace(sysRootObject, bounce, thePayload);
        thePayload.radiance = varAttenuation * thePayload.radiance;
    } else {
        thePayload.radiance = optix::make_float3(0);
    }
}

RT_PROGRAM void any_hit() {}
