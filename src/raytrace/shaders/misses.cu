#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "payload.h"

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayPayload, thePayload, rtPayload, );
rtDeclareVariable(optix::float3, sysColorSky, , );
rtDeclareVariable(optix::float3, sysColorGround, , );

RT_PROGRAM void miss_gradient(void) {
    const float t       = theRay.direction.y * 0.5f + 0.5f;
    thePayload.radiance = optix::lerp(sysColorGround, sysColorSky, t);
}
