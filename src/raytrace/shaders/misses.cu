#include "hip/hip_runtime.h"
#include "config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "payload.h"

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

rtDeclareVariable(RayPayload, thePayload, rtPayload, );

RT_PROGRAM void miss_gradient(void) {
    const float t       = theRay.direction.y * 0.5f + 0.5f;
    thePayload.radiance = optix::lerp(make_float3(0.0f), make_float3(1.0f), t);
}
