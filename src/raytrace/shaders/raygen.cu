#include "hip/hip_runtime.h"
#include "config.h"

#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

#include "payload.h"

// Note, the nomenclature used in the device code of all optixIntroduction samples
// follows some simple rules using prefixes to help indicating the scope and meaning:
//
// "sys" = renderer "system"-wide variables, defined at global context scope.
// "the" = variables with OptiX built-in semantic, like rtLaunchIndex, etc.
// "var" = "varyings" with developer defined attribute semantic, calculated by the intersection
// program. "par" = "parameter" variable held at some object scope, not at the global context scope.
//         (Exception to the last rule are the vertex "attributes" and "indices" held at Geometry
//         nodes.)

rtBuffer<uchar4, 2> sysOutputBuffer;  // RGBA

rtDeclareVariable(rtObject, sysRootObject, , );

rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

// exposure + simple Reinhard tonemapper + gamma
RT_FUNCTION uchar4 display(float3 val) {
    // clamp
    // val.x = fminf(fmaxf(val.x, 0.0f), 1.0f);
    // val.y = fminf(fmaxf(val.y, 0.0f), 1.0f);
    // val.z = fminf(fmaxf(val.z, 0.0f), 1.0f);

    // do tonemap
    // val *= 1.0f;
    // const float burn_out = 0.1f;
    // val.x *= (1.0f + val.x * burn_out) / (1.0f + val.x);
    // val.y *= (1.0f + val.y * burn_out) / (1.0f + val.y);
    // val.z *= (1.0f + val.z * burn_out) / (1.0f + val.z);
    // // these WERE in z->y->x order
    // return make_uchar4(
    //     (unsigned char)(255.0 * fminf(powf(fmaxf(val.x, 0.0f), (float)(1.0 / 2.2)), 1.0f)),
    //     (unsigned char)(255.0 * fminf(powf(fmaxf(val.y, 0.0f), (float)(1.0 / 2.2)), 1.0f)),
    //     (unsigned char)(255.0 * fminf(powf(fmaxf(val.z, 0.0f), (float)(1.0 / 2.2)), 1.0f)), 255);

    // do bt_709
    uchar4 output = make_uchar4(0u, 255u, 0u, 255u);
    if (val.x < 0.018) {
        output.x = (unsigned char)(val.x * 4.5 * 255.0);
    } else {
        output.x = (unsigned char)(255.0 * (1.099 * pow(val.x, 0.45) - 0.099));
    }
    if (val.y < 0.018) {
        output.y = (unsigned char)(val.y * 4.5 * 255.0);
    } else {
        output.y = (unsigned char)(255.0 * (1.099 * pow(val.y, 0.45) - 0.099));
    }
    if (val.z < 0.018) {
        output.z = (unsigned char)(val.z * 4.5 * 255.0);
    } else {
        output.z = (unsigned char)(255.0 * (1.099 * pow(val.z, 0.45) - 0.099));
    }

    return output;
}

// Entry point
RT_PROGRAM void raygen() {
    RayPayload payload;

    payload.radiance = make_float3(0.0f);

    // The launch index is the pixel coordinate.
    // Note that launchIndex = (0, 0) is the bottom left corner of the image.
    const float2 pixel = make_float2(theLaunchIndex);

    // sample the ray in the center of the pixel.
    const float2 fragment = pixel + make_float2(0.5f);

    const float2 screen = make_float2(theLaunchDim);

    // normalized device coordinates in range [-1, 1].
    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    const float3 origin = sysCameraPosition;

    // must be normalized
    const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

    // start the ray traversal at the root object
    rtTrace(sysRootObject, ray, payload);

    sysOutputBuffer[theLaunchIndex] = display(payload.radiance);
}
