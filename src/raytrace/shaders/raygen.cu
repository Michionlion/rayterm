#include "hip/hip_runtime.h"
#include "config.h"

#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

#include "payload.h"

// Note, the nomenclature used in the device code of all optixIntroduction samples
// follows some simple rules using prefixes to help indicating the scope and meaning:
//
// "sys" = renderer "system"-wide variables, defined at global context scope.
// "the" = variables with OptiX built-in semantic, like rtLaunchIndex, etc.
// "var" = "varyings" with developer defined attribute semantic, calculated by the intersection
// program. "par" = "parameter" variable held at some object scope, not at the global context scope.
//         (Exception to the last rule are the vertex "attributes" and "indices" held at Geometry
//         nodes.)

rtBuffer<uchar4, 2> sysOutputBuffer;  // RGBA

rtDeclareVariable(rtObject, sysRootObject, , );

rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

// Entry point
RT_PROGRAM void raygen() {
    RayPayload payload;

    payload.radiance = make_float3(0.0f);

    // The launch index is the pixel coordinate.
    // Note that launchIndex = (0, 0) is the bottom left corner of the image.
    const float2 pixel = make_float2(theLaunchIndex);

    // sample the ray in the center of the pixel.
    const float2 fragment = pixel + make_float2(0.5f);

    const float2 screen = make_float2(theLaunchDim);

    // normalized device coordinates in range [-1, 1].
    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    const float3 origin = sysCameraPosition;

    // must be normalized
    const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

    // start the ray traversal at the root object
    rtTrace(sysRootObject, ray, payload);

    unsigned char x, y, z;
    if (payload.radiance.x < 0.018) {
        x = (unsigned char)(payload.radiance.x * 4.5 * 255.0);
    } else {
        x = (unsigned char)(255.0 * (1.099 * pow(payload.radiance.x, 0.45) - 0.099));
    }
    if (payload.radiance.y < 0.018) {
        y = (unsigned char)(payload.radiance.y * 4.5 * 255.0);
    } else {
        y = (unsigned char)(255.0 * (1.099 * pow(payload.radiance.y, 0.45) - 0.099));
    }
    if (payload.radiance.z < 0.018) {
        z = (unsigned char)(payload.radiance.z * 4.5 * 255.0);
    } else {
        z = (unsigned char)(255.0 * (1.099 * pow(payload.radiance.z, 0.45) - 0.099));
    }

    sysOutputBuffer[theLaunchIndex] = make_uchar4(x, y, z, (unsigned char)(255));
}
