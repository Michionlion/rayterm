#include "hip/hip_runtime.h"
#include "config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "payload.h"

// Note, the nomenclature used in the device code of all optixIntroduction samples
// follows some simple rules using prefixes to help indicating the scope and meaning:
//
// "sys" = renderer "system"-wide variables, defined at global context scope.
// "the" = variables with OptiX built-in semantic, like rtLaunchIndex, etc.
// "var" = "varyings" with developer defined attribute semantic, calculated by the intersection
// program. "par" = "parameter" variable held at some object scope, not at the global context scope.
//         (Exception to the last rule are the vertex "attributes" and "indices" held at Geometry
//         nodes.)

rtBuffer<float4, 2> sysOutputBuffer;  // RGBA32F

rtDeclareVariable(rtObject, sysRootObject, , );

rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

// rtDeclareVariable(float3, sysCameraPosition, , );
// rtDeclareVariable(float3, sysCameraU, , );
// rtDeclareVariable(float3, sysCameraV, , );
// rtDeclareVariable(float3, sysCameraW, , );

// Entry point for a pinhole camera.
RT_PROGRAM void raygen() {
    RayPayload payload;

    payload.radiance = make_float3(0.0f);

    // The launch index is the pixel coordinate.
    // Note that launchIndex = (0, 0) is the bottom left corner of the image,
    // which matches the origin in the OpenGL texture used to display the result.
    const float2 pixel = make_float2(theLaunchIndex);
    // Sample the ray in the center of the pixel.
    const float2 fragment = pixel + make_float2(0.5f);
    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's
    // setup.
    const float2 screen = make_float2(theLaunchDim);
    // Normalized device coordinates in range [-1, 1].
    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    // const float3 origin    = sysCameraPosition;
    // const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV +
    // sysCameraW);
    const float3 origin    = make_float3(0.0f);
    const float3 direction = optix::normalize(make_float3(0.0f, ndc.y, -1.0f));

    // Shoot a ray from origin into direction (must always be normalized!) for ray type 0 and test
    // the interval between 0.0 and RT_DEFAULT_MAX for intersections. There is no geometry in the
    // scene, yet, so this will always invoke the miss program assigned to ray type 0, which is the
    // radiance ray in this implementation.
    optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

    // Start the ray traversal at the scene's root node, which in this case is an empty Group.
    // The ray becomes the variable with rtCurrentRay semantic in the other program domains.
    // The PerRayData becomes the variable with the semantic rtPayload in the other program domains,
    // which allows to exchange arbitrary data between the program domains.
    rtTrace(sysRootObject, ray, payload);

    sysOutputBuffer[theLaunchIndex] = make_float4(payload.radiance, 1.0f);
}
