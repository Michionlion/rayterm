#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "payload.h"
#include "random.h"

rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );

rtBuffer<float3> nbuf;
rtBuffer<int3> nidx;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayPayload, thePayload, rtPayload, );

RT_PROGRAM void closest_hit() {
    // calculate interpolated vertex (shading) normal
    const optix::int3 n_idx = nidx[rtGetPrimitiveIndex()];
    const optix::float3 normal =
        optix::normalize(nbuf[n_idx.y] * barycentrics.x + nbuf[n_idx.z] * barycentrics.y +
                         nbuf[n_idx.x] * (1.0f - barycentrics.x - barycentrics.y));
    thePayload.radiance = normal * 0.5f + 0.5f;
}

RT_PROGRAM void any_hit() {}
