#include "hip/hip_runtime.h"
#include <optix.h>
#include "config.h"

rtBuffer<uchar4, 2> sysOutputBuffer;

rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

RT_PROGRAM void exception() {
#if USE_DEBUG_EXCEPTIONS
    const unsigned int code = rtGetExceptionCode();
    if (RT_EXCEPTION_USER <= code) {
        rtPrintf("User exception %d at (%d, %d)\n", code - RT_EXCEPTION_USER, theLaunchIndex.x,
            theLaunchIndex.y);
    } else {
        rtPrintf("Exception code 0x%X at (%d, %d)\n", code, theLaunchIndex.x, theLaunchIndex.y);
    }
    // RGBA32F super magenta as error color (makes sure this isn't accumulated away in a progressive
    // renderer).
    sysOutputBuffer[theLaunchIndex] = make_uchar4(255, 0, 255, 255);
#endif
}
